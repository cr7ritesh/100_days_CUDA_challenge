
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void vecAddKernel(float* A, float* B, float* C, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) C[i] = B[i] + A[i];
}

int main() {
    int n = 10;

    float* A = new float[n];
    float* B = new float[n];
    float* C = new float[n];

    for(int i = 0; i < n; i++) {
        A[i] = i; B[i] = i * 2;
    }

    float *da, *db, *dc;

	hipMalloc((void**) &da, sizeof(float) * n);
    hipMalloc((void**) &db, sizeof(float) * n);
	hipMalloc((void**) &dc, sizeof(float) * n);

	hipMemcpy(da, A, sizeof(float) * n, hipMemcpyHostToDevice);
	hipMemcpy(db, B, sizeof(float) * n, hipMemcpyHostToDevice);

	vecAddKernel<<<ceil(n/256.0), 256>>>(da, db, dc, n);

	hipMemcpy(C, dc, sizeof(float) * n, hipMemcpyDeviceToHost);

	hipFree(da);hipFree(db);hipFree(dc);

    std::cout << "\nResults:\n";
    for(int i = 0; i < n; i++) {
        std::cout << A[i] << " + " << B[i] << " = " << C[i] << std::endl;
    }

    return 0;
}
