
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void RELU(float* a, float* b, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) b[i] = fmaxf(0.0f, a[i]);
}

int main() {
    int N = 10; 
    int size = N * sizeof(float);

    float* A = (float*)malloc(size);
    float* B = (float*)malloc(size);

    for (int i = 0; i < N; ++i) A[i] = float(i - 4); 

    float *da, *db;
    hipMalloc(&da, size);hipMalloc(&db, size);

    hipMemcpy(da, A, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = 1;

    RELU<<<gridSize, blockSize>>>(da, db, N);

    hipMemcpy(B, db, size, hipMemcpyDeviceToHost);

    std::cout << "Input: ";
    for (int i = 0; i < N; ++i) std::cout << A[i] << " ";
    std::cout << std::endl;

    std::cout << "Output: ";
    for (int i = 0; i < N; ++i) std::cout << B[i] << " ";

    hipFree(da);hipFree(db);

    return 0;
}